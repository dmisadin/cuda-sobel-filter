#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <chrono>
#include ""
using namespace cv;
using namespace std;

// CUDA kernel for grayscale conversion
__global__ void rgb2gray(const uchar3* input, unsigned char* gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;
    uchar3 pixel = input[idx];
    gray[idx] = static_cast<unsigned char>(0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z);
}

// CUDA kernel for Sobel filter (X direction only for simplicity)
__global__ void sobelFilter(const unsigned char* gray, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= 0 || x >= width - 1 || y <= 0 || y >= height - 1) return;

    int idx = y * width + x;

    int gx = - gray[(y - 1) * width + (x - 1)]
             - 2 * gray[y * width + (x - 1)]
             - gray[(y + 1) * width + (x - 1)]
             + gray[(y - 1) * width + (x + 1)]
             + 2 * gray[y * width + (x + 1)]
             + gray[(y + 1) * width + (x + 1)];

    int gy = - gray[(y - 1) * width + (x - 1)]
             - 2 * gray[(y - 1) * width + x]
             - gray[(y - 1) * width + (x + 1)]
             + gray[(y + 1) * width + (x - 1)]
             + 2 * gray[(y + 1) * width + x]
             + gray[(y + 1) * width + (x + 1)];

    gx = abs(gx);
    gy = abs(gy);

    int magnitude = sqrtf((float)(gx * gx + gy * gy));

    output[idx] = magnitude > 255 ? 255 : magnitude;
}

void sobelFilterCPU(const cv::Mat& gray, cv::Mat& output) {
    int width = gray.cols;
    int height = gray.rows;

    output = cv::Mat::zeros(height, width, CV_8U);

    for (int y = 1; y < height - 1; ++y) {
        for (int x = 1; x < width - 1; ++x) {
            int gx = - gray.at<uchar>(y - 1, x - 1)
                     - 2 * gray.at<uchar>(y, x - 1)
                     - gray.at<uchar>(y + 1, x - 1)
                     + gray.at<uchar>(y - 1, x + 1)
                     + 2 * gray.at<uchar>(y, x + 1)
                     + gray.at<uchar>(y + 1, x + 1);

            int gy = - gray.at<uchar>(y - 1, x - 1)
                     - 2 * gray.at<uchar>(y - 1, x)
                     - gray.at<uchar>(y - 1, x + 1)
                     + gray.at<uchar>(y + 1, x - 1)
                     + 2 * gray.at<uchar>(y + 1, x)
                     + gray.at<uchar>(y + 1, x + 1);

            int magnitude = sqrtf((float)(gx * gx + gy * gy));

            // saturacija
            output.at<uchar>(y, x) = magnitude > 255 ? 255 : magnitude;
        }
    }
}

int main() {
    // Ucitaj RGB sliku
    Mat image = imread("large_image.jpg", IMREAD_COLOR);
    if (image.empty()) {
        cerr << "Greska: Slika nije ucitana!" << endl;
        return -1;
    }

    int width = image.cols;
    int height = image.rows;
    cout << "Dimenzije slike: " << width << "x" << height << endl;

    // ---------- CPU SLOBEL ----------
    Mat gray_cpu, sobel_cpu;
    auto t1 = chrono::high_resolution_clock::now();
    cvtColor(image, gray_cpu, COLOR_BGR2GRAY);
    sobelFilterCPU(gray_cpu, sobel_cpu);
    auto t2 = chrono::high_resolution_clock::now();
    cout << "CPU vrijeme: " << chrono::duration_cast<chrono::microseconds>(t2 - t1).count() << " us" << endl;

    // ---------- GPU SOBEL ----------
    uchar3* d_input;
    unsigned char* d_gray, * d_output;
    size_t numPixels = width * height;

    // Alokacija memorije na GPU
    hipMalloc(&d_input, numPixels * sizeof(uchar3));
    hipMalloc(&d_gray, numPixels * sizeof(unsigned char));
    hipMalloc(&d_output, numPixels * sizeof(unsigned char));

    // Kopiraj podatke na GPU
    hipMemcpy(d_input, image.ptr<uchar3>(), numPixels * sizeof(uchar3), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);

    auto t3 = chrono::high_resolution_clock::now();
    rgb2gray << <grid, block >> > (d_input, d_gray, width, height);
    hipDeviceSynchronize();

    sobelFilter << <grid, block >> > (d_gray, d_output, width, height);
    hipDeviceSynchronize();
    auto t4 = chrono::high_resolution_clock::now();
    cout << "GPU vrijeme: " << chrono::duration_cast<chrono::microseconds>(t4 - t3).count() << " us" << endl;

    // Prebaci rezultat natrag
    Mat result_gpu(height, width, CV_8U);
    hipMemcpy(result_gpu.ptr(), d_output, numPixels * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Oslobodi memoriju
    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_output);

    // Spremi i prikazi rezultate
    imwrite("sobel_cpu.jpg", sobel_cpu);
    imwrite("sobel_gpu.jpg", result_gpu);
    cout << "Rezultati spremljeni." << endl;

    return 0;
}